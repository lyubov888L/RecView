#include "hip/hip_runtime.h"
#if !defined( _CUDARECONST_CU_ )
#define _CUDARECONST_CU_

#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
//#include "cudaFFT.cu"

////////////////////////
//Tomographic reconstruction routines
typedef int BOOL;

#include "reconstinfo.h"
#include "cudaReconst.h"
#include "constants.h"
//#include "sm_12_atomic_functions.h"

texture<int, 1, hipReadModeElementType> tex_igp;
int blocksize = CUDA_BLOCKSIZE;


//160804 kernel4 was rrevised but not tested
__global__ void
projKernel4(int* d_ifp, int ixdimp, float fsin, float fcos, float foffset) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	int ixy = ix;
	for (int iy=0; iy<ixdimp; iy++) {
		float fx1 = ix * fcos + iy * fsin + foffset; 
		d_ifp[ixy] += tex1Dfetch(tex_igp, (int)(fx1));
		ixy += ixdimp;
	}
}

__global__ void
projKernel8f(int* d_ifp, int ixdimp, float fsin, float fcos, float foffset) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	int iy = blockIdx.y << 1;
	float fx1 = ix * fcos + iy * fsin + foffset; 
	int ixy = ix + ixdimp * iy;
	d_ifp[ixy] += tex1Dfetch(tex_igp, (int)(fx1));
	if (iy >= ixdimp - 1) return;
	d_ifp[ixy + ixdimp] += tex1Dfetch(tex_igp, (int)(fx1 + fsin));
}

__global__ void
p2igpCopyKernel(float2* d_p, int* d_igp, int ixdimp, int ihoffset, float fscale) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	//without DBPT_GINTP:
	//d_igp[ix] = (int)(d_p[ix + ihoffset].x * fscale);
	//for (int j=0; j<ixdimp; j++) {d_igp[j] = (int)(d_p[j + ihoffset].x * fscale);}
	//
	//with DBPT_GINTP
	const int gidx = ix * DBPT_GINTP;
	int iy = blockIdx.y;
	if (iy == 0) {d_igp[gidx] = (int)(d_p[ix + ihoffset].x * fscale); return;}
	const float p1p0 = (ix == ixdimp -1)? 
		0.0f : (d_p[ix + ihoffset + 1].x - d_p[ix + ihoffset].x) / DBPT_GINTP * fscale;
	d_igp[gidx + iy] = (int)(d_p[ix + ihoffset].x * fscale + p1p0 * iy);
	//
	//for (int j=0; j<ixdimp; j++) {
	//	const TCmpElmnt p0 = p[j + ihoffset].re * BACKPROJ_SCALE;
	//	const TCmpElmnt p1p0 = (j == ixdimp -1)? 
	//		0.0f : (p[j + ihoffset + 1].re - p[j + ihoffset + 1].re) / DBPT_GINTP * BACKPROJ_SCALE;
	//	const int gidx = (j + imargin) * DBPT_GINTP;
	//	for (int k=0; k<DBPT_GINTP; k++) {igp[gidx + k] = (int)(p0 + p1p0 * k);}
	//}
}

__global__ void
filtKernel(float2* d_p, float* d_filt, int ndim) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ndim) return;
	d_p[ix].x *= d_filt[ix];
	d_p[ix].y *= d_filt[ix];
	//for (int k=0; k<ndim; k++) {d_p[k].x *= d_filt[k]; d_p[k].y *= d_filt[k];}
}

__global__ void
intpKernel(float2* d_p, short* d_strip, int ixdim, int ndim, int iIntpDim, float center) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdim) return;
	int idx = (ix - (int)center) * iIntpDim + (ndim / 2);
	if (idx < 0) return;
	if (idx >= ndim) return;
	int iy = blockIdx.y;
	if (iy == 0) {d_p[idx].x = d_strip[ix]; return;}
	//interpolation
	if (ix == ixdim - 1) return;
	d_p[idx+iy].x = (float)(d_strip[ix] * (iIntpDim - iy) + d_strip[ix+1] * iy) / iIntpDim;
	/*
	for (int k=0; k<ixdim; k++) {
		int idx = (k - (int)center) * iIntpDim + (ndim / 2);
		if (idx < 0) continue;
		if (idx >= ndim) break;
		d_p[idx].x = d_strip[k];
		//interpolation
		if (k == ixdim - 1) break;
		for (int j=1; j<iIntpDim; j++) {
			d_p[idx+j].x = (float)(d_strip[k] * (iIntpDim - j) / iIntpDim + d_strip[k+1] * j / iIntpDim);
		}
	}*/
}

extern "C" 
void CudaDeconvBackProj(int ixdim, int iIntpDim, int ndim, float center, float theta,  
				int* d_ifp, float* d_filt, short* d_strip, int* d_igp, float2* d_p, hipfftHandle* fftplan) {
	//constants
	const int ixdimp = ixdim * iIntpDim;
	const int ixdimh = ixdimp / 2;
	const int ihoffset = ndim / 2 - ixdimh;
	//
	//kernel parameters
	//const int blocksize = CUDA_BLOCKSIZE;// ==> blockDim.x;
    dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil( ixdimp / (float)blocksize));
	dim3 dimGrid_ixdimp(gridsize);//0<=blockIdx.x<gridsize
	//igp texture
	textureReference* texRefPtr;
	//131011 hipGetTextureReference((const textureReference **)&texRefPtr, "tex_igp");
	if (hipSuccess != hipGetTextureReference((const textureReference **)&texRefPtr, &tex_igp)) return;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	//with DBPT_GINTP
	const unsigned int mem_size_igp = sizeof(int) * ixdimp * DBPT_GINTP;
	//without DBPT_GINTP
    //const unsigned int mem_size_igp = sizeof(int) * ixdimp;
	hipBindTexture(0, texRefPtr, d_igp, &channelDesc, mem_size_igp);
	//130923 cutilSafeCall( hipBindTexture(0, texRefPtr, d_igp, &channelDesc, mem_size_igp));
	//090312 hipBindTexture(0, texRefPtr, d_igp, &channelDesc, mem_size_igp);
	tex_igp.addressMode[0] = hipAddressModeClamp;
	tex_igp.filterMode = hipFilterModePoint;
	tex_igp.normalized = false;    // access with integer texture coordinates
	//
	//interpolation
    const unsigned int mem_size_p = sizeof(float2) * ndim;
	hipMemset(d_p, 0, mem_size_p);
	//130923 cutilSafeCall(hipMemset(d_p, 0, mem_size_p) );
	//090312 hipMemset(d_p, 0, mem_size_p);
	const int gridsize_intp = (int)(ceil( ixdim / (float)blocksize));
	dim3 dimGrid_intp(gridsize_intp, iIntpDim);
	intpKernel<<< dimGrid_intp, dimBlock >>>(d_p, d_strip, ixdim, ndim, iIntpDim, center);
	hipDeviceSynchronize();
	//FFT-filter
	hipfftExecC2C(*fftplan, (hipfftComplex*)d_p, (hipfftComplex*)d_p, HIPFFT_FORWARD );
	//
	const int gridsize_ndim = (int)(ceil( ndim / (float)blocksize));
	dim3 dimGrid_ndim(gridsize_ndim);
	filtKernel<<< dimGrid_ndim, dimBlock >>>(d_p, d_filt, ndim);
	hipDeviceSynchronize();
	//
	hipfftExecC2C(*fftplan, (hipfftComplex*)d_p, (hipfftComplex*)d_p, HIPFFT_BACKWARD );
	//
	float fscale = (float)BACKPROJ_SCALE / ndim;
	dim3 dimGrid_p2igp(gridsize, DBPT_GINTP);
	p2igpCopyKernel<<< dimGrid_p2igp, dimBlock >>>(d_p, d_igp, ixdimp, ihoffset, fscale);
	//without DBPT_GINTP
	//p2igpCopyKernel<<< dimGrid_ixdimp, dimBlock >>>(d_p, d_igp, ixdimp, ihoffset, fscale);
	hipDeviceSynchronize();
	//params
	const float fcos = (float)(cos(theta) * DBPT_GINTP);
	const float fsin = (float)(-sin(theta) * DBPT_GINTP);
	const float fcenter = (float)((ixdimh + center - (int)(center)) * DBPT_GINTP);
	const float foffset = fcenter - ixdimh * (fcos + fsin);
	//Kernels
	//shared memory must not be used since its use causes device driver crash
	//
	//Kernel4 for emulation mode
	//projKernel4<<< dimGrid_ixdimp, dimBlock >>>(d_ifp, ixdimp, fsin, fcos, foffset);
	//
	//Kernel8f
	int iydim = (ixdimp >> 1) + (ixdimp & 0x01);
	dim3 dimGrid(gridsize, iydim);// 0<=blockIdx.x<gridsize, 0<=blockIdx.y<iydim
	projKernel8f<<< dimGrid, dimBlock >>>(d_ifp, ixdimp, fsin, fcos, foffset);
	//
	//hipDeviceSynchronize();
}

extern "C" 
void CudaBackProj(int ixdim, int iIntpDim, float center, float theta, int* d_ifp, int* d_igp) {
	//constants
	const int ixdimp = ixdim * iIntpDim;
	const int ixdimh = ixdimp / 2;
	//
	//kernel parameters
	//const int blocksize = CUDA_BLOCKSIZE;// ==> blockDim.x;
    dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil( ixdimp / (float)blocksize));
	//igp texture
	textureReference* texRefPtr;
	//131011 if (hipSuccess != hipGetTextureReference((const textureReference **)&texRefPtr, "tex_igp")) return;
	if (hipSuccess != hipGetTextureReference((const textureReference **)&texRefPtr, &tex_igp)) return;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	//with DBPT_GINTP
	const unsigned int mem_size_igp = sizeof(int) * ixdimp * DBPT_GINTP;
	if (hipSuccess != hipBindTexture(0, texRefPtr, d_igp, &channelDesc, mem_size_igp)) return;
	//130923 cutilSafeCall( hipBindTexture(0, texRefPtr, d_igp, &channelDesc, mem_size_igp));
	tex_igp.addressMode[0] = hipAddressModeClamp;
	tex_igp.filterMode = hipFilterModePoint;
	tex_igp.normalized = false;    // access with integer texture coordinates
	//params
	const float fcos = (float)(cos(theta) * DBPT_GINTP);
	const float fsin = (float)(-sin(theta) * DBPT_GINTP);
	const float fcenter = (float)((ixdimh + center - (int)(center)) * DBPT_GINTP);
	const float foffset = fcenter - ixdimh * (fcos + fsin);
	//
	//Kernel4 for emulation mode
	//dim3 dimGrid_ixdimp(gridsize);
	//projKernel4<<< dimGrid_ixdimp, dimBlock >>>(d_ifp, ixdimp, fsin, fcos, foffset);
	//
	//Kernel8f
	int iydim = (ixdimp >> 1) + (ixdimp & 0x01);
	dim3 dimGrid(gridsize, iydim);// 0<=blockIdx.x<gridsize, 0<=blockIdx.y<iydim
	projKernel8f<<< dimGrid, dimBlock >>>(d_ifp, ixdimp, fsin, fcos, foffset);
}

__global__ void
sinoKernel(short* d_Dark, short* d_Incident, short* d_Strip, int ixmul, float t0) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixmul) return;
	if (d_Incident[ix] <= 0) {d_Strip[ix] = 0; return;}
	int iSample = d_Strip[ix] - d_Dark[ix];
	if (iSample < SINOGRAM_PIXEL_MIN) {d_Strip[ix] = 0; return;}
	d_Strip[ix] = (short)(log(d_Incident[ix] / (double)iSample) * LOG_SCALE + 0.5);
	//for (int j=0; j<ixmul; j++) {
	//	int d_Incident = d_Incident0[j] + (int)((d_Incident1[j] - d_Incident0[j] + 0.5) * t0) - d_Dark[j];
	//	if (d_Incident <= 0) {d_Strip[j] = 0; continue;}
	//	int iSample = d_Strip[j] - d_Dark[j];
	//	if (iSample < SINOGRAM_PIXEL_MIN) {d_Strip[j] = 0; continue;}
	//	d_Strip[j] = (short)(log((double)d_Incident / (double)iSample) * LOG_SCALE + 0.5);
	//}
}

extern "C" 
void CudaSinogram(short* d_Strip, int ixmul, float t0) {
    dim3 dimBlock(blocksize, 1);
	const int gridsize_ixmul = (int)(ceil( ixmul / (float)blocksize));
	dim3 dimGrid_ixmul(gridsize_ixmul);
	short* d_Incident = &(d_Strip[ixmul]);
	short* d_Dark = &(d_Strip[ixmul * 2]);
	//normal
	sinoKernel<<< dimGrid_ixmul, dimBlock >>>(d_Dark, d_Incident, d_Strip, ixmul, t0);
}

__global__ void
lsqfitKernel(short* d_ref, short* d_qry, int ixref, int iyref, int ixqry, int iyqry, int ix, int iy, 
					unsigned __int64* d_result) {
	int jrx = blockDim.x * blockIdx.x + threadIdx.x;
	if (jrx >= ixref) return;
	const int jqx = jrx + ix;
	if ((jqx < 0)||(jqx >= ixqry)) return;
	//unsigned __int64 nlsqsum = 0;
	int nlsqsum = 0;
	unsigned __int64 ilsqsum = 0;
	for (int jry=0; jry<iyref; jry++) {
		const int jqy = jry + iy;
		if ((jqy < 0)||(jqy >= iyqry)) continue;
		int dr = d_ref[jry * ixref + jrx];
		if (dr == SHRT_MIN) continue;
		int dq = d_qry[jqy * ixqry + jqx];
		if (dq == SHRT_MIN) continue;
		//130207 unsigned __int64 idiff = dr - dq;
		__int64 idiff = dr - dq;
		ilsqsum += idiff * idiff;
		nlsqsum++;
	}
	d_result[jrx] = ilsqsum;
	d_result[jrx + ixref] = nlsqsum;
	//atomicAdd(&(d_result[0]), ilsqsum);
	//atomicAdd(&(d_result[1]), nlsqsum);
}

extern "C" 
void CudaLsqfit(short* d_ref, short* d_qry, int ixref, int iyref, int ixqry, int iyqry,
					int ix, int iy, unsigned __int64* d_result) {
    dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil( ixref / (float)blocksize));
	dim3 dimGrid(gridsize);
	lsqfitKernel<<< dimGrid, dimBlock >>>(d_ref, d_qry, ixref, iyref, ixqry, iyqry, ix, iy, d_result);
}

extern "C" int GetCudaDeviceCount() {
    int deviceCount;
	hipError_t cerr = hipGetDeviceCount(&deviceCount);
	if (cerr == hipErrorNoDevice) {
		return 0;
	} else if (cerr == hipErrorInsufficientDriver) {
		return CUDA_ERROR_INSUFFICIENT_DRIVER;
	} else if (cerr != hipSuccess) {
		return 0;
	}
	//130923 cutilSafeCall(hipGetDeviceCount(&deviceCount));

    //detect virtual device
    if (deviceCount) {
	    hipDeviceProp_t deviceProp;
		if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) return 0;
		//130923 cutilSafeCall(hipGetDeviceProperties(&deviceProp, 0));
	    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
			//comment out the follwing line to enable virtual device
			deviceCount = 0;
		}
	}
    return deviceCount;
}

extern "C" int GetCudaMaxThreadsPerBlock() {
	int iCUDAblock = 65536;
    int iDeviceCount = GetCudaDeviceCount();
	if (iDeviceCount == CUDA_ERROR_INSUFFICIENT_DRIVER) iDeviceCount = 0;
    //detect virtual device
	hipDeviceProp_t deviceProp;
    if (iDeviceCount) {
		if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) return CUDA_BLOCKSIZE;
		//130923 cutilSafeCall(hipGetDeviceProperties(&deviceProp, 0));
	    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
		    iDeviceCount = 0; iCUDAblock = CUDA_BLOCKSIZE;
		}
		for (int i=0; i<iDeviceCount; i++) {
			if (deviceProp.maxThreadsPerBlock < iCUDAblock) iCUDAblock = deviceProp.maxThreadsPerBlock;
		}
	}
	return iCUDAblock;
}

extern "C" int GetCudaWarpSize() {
	int iCUDAwarp = CUDA_WARPSIZE;
    int iDeviceCount = GetCudaDeviceCount();
	if (iDeviceCount == CUDA_ERROR_INSUFFICIENT_DRIVER) iDeviceCount = 0;
    //detect virtual device
	hipDeviceProp_t deviceProp;
    if (iDeviceCount) {
		if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) return CUDA_WARPSIZE;
		//130923 cutilSafeCall(hipGetDeviceProperties(&deviceProp, 0));
	    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
		    iDeviceCount = 0;
		}
		for (int i=0; i<iDeviceCount; i++) {
			if (deviceProp.warpSize < iCUDAwarp) iCUDAwarp = deviceProp.warpSize;
		}
	}
	return iCUDAwarp;
}

/*
deviceQuery.cu

There is 1 device supporting CUDA

Device 0: "Quadro FX 3700"
  Major revision number:                         1
  Minor revision number:                         1
  Total amount of global memory:                 536870912 bytes
  Number of multiprocessors:                     14
  Number of cores:                               112
  Total amount of constant memory:               65536 bytes
  Total amount of shared memory per block:       16384 bytes
  Total number of registers available per block: 8192
  Warp size:                                     32
  Maximum number of threads per block:           512
  Maximum sizes of each dimension of a block:    512 x 512 x 64
  Maximum sizes of each dimension of a grid:     65535 x 65535 x 1
  Maximum memory pitch:                          262144 bytes
  Texture alignment:                             256 bytes
  Clock rate:                                    1.25 GHz
  Concurrent copy and execution:                 No

Test PASSED
*/

#endif //_CUDARECONST_CU_